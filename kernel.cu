
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

#define BLOCK_SIZE 512

__global__ void reduction(float *out, float *in, unsigned size)
{
    /********************************************************************
    Load a segment of the input vector into shared memory
    Traverse the reduction tree
    Write the computed sum to the output vector at the correct index
    ********************************************************************/

    // INSERT KERNEL CODE HERE

    //NaiveReductionKernal
    // __shared__ float partialSum[2*BLOCK_SIZE];

    // unsigned int t = threadIdx.x;
    // unsigned int start = 2 * blockIdx.x * blockDim.x;

    // if(start + t < size){
    //     partialSum[t] = in[start + t];
    // }else{
    //     partialSum[t] = 0.0;
    // }

    // if(start + blockDim.x + t < size){
    //     partialSum[blockDim.x + t] =in[start + blockDim.x + t];
    // }else{
    //     partialSum[blockDim.x + t] = 0.0;
    // }

    // for (unsigned int stride = 1; stride <= blockDim.x;  stride *= 2){
    // __syncthreads();
    //     if (t % stride == 0)
    //         partialSum[2*t]+= partialSum[2*t+stride];
            
    // }
    // if(t == 0)
    //     out[blockIdx.x] = partialSum[0];


    //better_version
    __shared__ float partialSum[2*BLOCK_SIZE];

    unsigned int t = threadIdx.x;
    unsigned int start = 2 * blockIdx.x * blockDim.x;

    if(start + t < size){
        partialSum[t] = in[start + t];
    }else{
        partialSum[t] = 0.0;
    }

    if(start + blockDim.x + t < size){
        partialSum[blockDim.x + t] =in[start + blockDim.x + t];
    }else{
        partialSum[blockDim.x + t] = 0.0;
    }
    __syncthreads();
    for (unsigned int stride = blockDim.x; stride > 0;  stride /= 2){
        
        if (t < stride)
            partialSum[t]+= partialSum[t+stride];
            __syncthreads();
    }
    if(t == 0)
        out[blockIdx.x] = partialSum[0];
}
